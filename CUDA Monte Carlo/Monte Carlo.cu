#include "hip/hip_runtime.h"
﻿
// CUDA Monte Carlo

// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

#ifndef DEBUG
#define DEBUG	false
#endif

#ifndef M_PI
#define M_PI	3.14f
#endif

// setting the number of trials in the monte carlo simulation:
#ifndef NUMTRIALS
#define NUMTRIALS 2048
#endif // !NUMTRIALS

#ifndef BLOCKSIZE
#define BLOCKSIZE		64	// number of threads in each block
#endif // !BLOCKSIZE

#ifndef NUMBLOCKS
#define NUMBLOCKS		NUMTRIALS/BLOCKSIZE
#endif

// ranges for the random numbers:
const float GMIN = 20.0;	// ground distance in meters
const float GMAX = 30.0;	// ground distance in meters
const float HMIN = 10.0;	// cliff height in meters
const float HMAX = 40.0;	// cliff height in meters
const float DMIN = 10.0;	// distance to castle in meters
const float DMAX = 20.0;	// distance to castle in meters
const float VMIN = 30.0;	// intial cnnonball velocity in meters / sec
const float VMAX = 50.0;	// intial cnnonball velocity in meters / sec
const float THMIN = 70.0;	// cannonball launch angle in degrees
const float THMAX = 80.0;	// cannonball launch angle in degrees

__constant__ float GRAVITY = -9.8;	// acceleraion due to gravity in meters / sec^2

__constant__ float TOL = 5.0;		// tolerance in cannonball hitting the castle in meters
				// castle is destroyed if cannonball lands between d-TOL and d+TOL

void
CudaCheckError()
{
	hipError_t e = hipGetLastError();
	if (e != hipSuccess)
	{
		fprintf(stderr, "CUDA failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));
	}
}

void
TimeOfDaySeed()
{
	struct tm y2k = { 0 };
	y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
	y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

	time_t  timer;
	time(&timer);
	double seconds = difftime(timer, mktime(&y2k));
	unsigned int seed = (unsigned int)(1000. * seconds);    // milliseconds
	srand(seed);
}

float
Ranf(float low, float high)
{
	float r = (float)rand();               // 0 - RAND_MAX
	float t = r / (float)RAND_MAX;       // 0. - 1.

	return   low + t * (high - low);
}

int
Ranf(int ilow, int ihigh)
{
	float low = (float)ilow;
	float high = ceil((float)ihigh);

	return (int)Ranf(low, high);
}

// degrees-to-radians -- callable from the device:
__device__
float
Radians(float d)
{
	return (M_PI / 180.f) * d;
}

// the kernel:
__global__
void
MonteCarlo(float* dvs, float* dths, float* dgs, float* dhs, float* dds, int* dhits)
{
	unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;

	// randomize everything:
	float v = dvs[gid];
	float thr = Radians(dths[gid]);
	float vx = v * cos(thr);
	float vy = v * sin(thr);
	float  g = dgs[gid];
	float  h = dhs[gid];
	float  d = dds[gid];

	int numHits = 0;

	// see if the ball doesn't even reach the cliff:
	float t = -vy / (0.5 * GRAVITY);
	float x = vx * t;
	if (x > g) {
		t = g / vx;
		float y = vy * t + 0.5 * GRAVITY * t * t;
		if (y > h)
		{
			float a = 0.5 * GRAVITY;
			float b = vy;
			float c = -h;
			float disc = b * b - 4.f * a * c;

			if (disc > 0.) {
				// successfully hits the ground above the cliff:
				// get the intersection:
				disc = sqrtf(disc);
				float t1 = (-b + disc) / (2.f * a);	// time to intersect high ground
				float t2 = (-b - disc) / (2.f * a);	// time to intersect high ground

				// only care about the second intersection
				float tmax = t1;
				if (t2 > t1)
					tmax = t2;

				// how far does the ball land horizontlly from the edge of the cliff?
				float upperDist = vx * tmax - g;

				// see if the ball hits the castle:
				if (fabs(upperDist - d) < TOL)
				{
					numHits += 1;
				}
			}
		}
	}
	dhits[gid] = numHits;
}

// these two #defines are just to label things
// other than that, they do nothing:
#define IN
#define OUT

// main program:

int
main(int argc, char* argv[])
{
	FILE* file = fopen("project5.csv", "a");
	TimeOfDaySeed();

	int dev = findCudaDevice(argc, (const char**)argv);

	// better to define these here so that the rand() calls don't get into the thread timing:
	float* hvs = new float[NUMTRIALS];
	float* hths = new float[NUMTRIALS];
	float* hgs = new float[NUMTRIALS];
	float* hhs = new float[NUMTRIALS];
	float* hds = new float[NUMTRIALS];
	int* hhits = new int[NUMTRIALS];

	// fill the random-value array:

	for (int n = 0; n < NUMTRIALS; n++)
	{
		hvs[n] = Ranf(VMIN, VMAX);
		hths[n] = Ranf(THMIN, THMAX);
		hgs[n] = Ranf(GMIN, GMAX);
		hhs[n] = Ranf(HMIN, HMAX);
		hds[n] = Ranf(DMIN, DMAX);
	}

	// allocate device memory:

	float *dvs, *dths, *dgs, *dhs, *dds;
	int *dhits;

	hipMalloc(&dvs, NUMTRIALS * sizeof(float));
	hipMalloc(&dths, NUMTRIALS * sizeof(float));
	hipMalloc(&dgs, NUMTRIALS * sizeof(float));
	hipMalloc(&dhs, NUMTRIALS * sizeof(float));
	hipMalloc(&dds, NUMTRIALS * sizeof(float));
	hipMalloc(&dhits, NUMTRIALS * sizeof(int));
	CudaCheckError();

	// copy host memory to the device:

	hipMemcpy(dvs, hvs, NUMTRIALS * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dths, hths, NUMTRIALS * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dgs, hgs, NUMTRIALS * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dhs, hhs, NUMTRIALS * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dds, hds, NUMTRIALS * sizeof(float), hipMemcpyHostToDevice);
	CudaCheckError();

	// setup the execution parameters:

	dim3 grid(NUMBLOCKS, 1, 1);
	dim3 threads(BLOCKSIZE, 1, 1);

	// allocate the events that we'll use for timing:

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	CudaCheckError();

	// create and start the timer:

	hipDeviceSynchronize();

	// record the start event:

	hipEventRecord(start, NULL);
	CudaCheckError();

	// execute the kernel:

	MonteCarlo <<< grid, threads >>> (IN dvs, IN dths, IN dgs, IN dhs, IN dds, OUT dhits);

	// record the stop event:

	hipEventRecord(stop, NULL);
	CudaCheckError();

	// wait for the stop event to complete:
	hipDeviceSynchronize();
	hipEventSynchronize(stop);
	CudaCheckError();

	float msecTotal;
	hipEventElapsedTime(&msecTotal, start, stop);
	CudaCheckError();

	// compute and print the performance

	double secondsTotal = 0.001 * (double)msecTotal;
	double megaTrialsPerSecond = (double)NUMTRIALS / secondsTotal / 1000000.;

	// fprintf(stderr, "NUMTRIALS = %d , BLOCKSIZE = %d , megaTrials/Sec = %lf\n", NUMTRIALS, BLOCKSIZE, megaTrialsPerSecond);

	// copy result from the device to the host:

	hipMemcpy(hhits, dhits, NUMTRIALS * sizeof(int), hipMemcpyDeviceToHost);
	CudaCheckError();

	// add up the hhits[ ] array: :
	
	int totalhits = 0;

	for (int i = 0; i < NUMTRIALS; i++)
	{
		totalhits += hhits[i];
	}

	// compute and print the probability:
	float probability = (float)totalhits / (float)NUMTRIALS;
	// fprintf(stderr, "probability = %lf \n", 100. * probability);
	fprintf(file, "%d,%d,%lf,%lf,%lf\n", NUMTRIALS, BLOCKSIZE, megaTrialsPerSecond, 100. * probability);
	// clean up host memory:
	delete[] hvs;
	delete[] hths;
	delete[] hgs;
	delete[] hhs;
	delete[] hds;
	delete[] hhits;


	// clean up device memory:
	hipFree(dvs);
	hipFree(dths);
	hipFree(dgs);
	hipFree(dhs);
	hipFree(dds);
	hipFree(dhits);
	CudaCheckError();

	return 0;
}

